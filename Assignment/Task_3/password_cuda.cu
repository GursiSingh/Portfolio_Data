//TASK 3

/*
Generate encrypted password in the kernel function (using CudaCrypt function) to be 
compared to original encrypted password (25 marks)
Allocating the correct amount of memory on the GPU based on input data. Memory is freed 
once used (15 marks)
Program works with multiple blocks and threads – the number of blocks and threads will 
depend on your kernel function. You will not be penalised if your program only works with a 
set number of blocks and threads however, your program must use more than one block (axis 
is up to you) and more than one thread (axis is up to you) (40 marks)
Decrypted password sent back to the CPU and printed (20 marks)

    Gursimran Singh -  2042387    
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


//This function gets two striings and return 0 if they match
__device__ int comparePasswords(const char *str_a, const char *str_b, unsigned len =256){

    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done){
    if ((str_a[i] == 0) || (str_b[i] == 0))
        done = 1;
    else if (str_a[i] != str_b[i]){
        match = i+1;
        if (((int)str_a[i] - (int)str_b[i]) < 0)
            match = 0 - (i + 1);
        }
        i++;
    }

    return match;
}

//This function encrypts the raw passsword and returns thee new encrypted password
__device__ char *cudaEncrypt(char *rawPassword,char *newPassword){
    
	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

//This function run over all the combinations of the password and rreturns the decrypted password
__global__ void cudaDecrypt(char* inputPassword, char *outputPassword, char* letterList, char* numList){

    char rawPassword[5];

    //Set current password
    rawPassword[0] = letterList[blockIdx.x];
    rawPassword[1] += letterList[blockIdx.y];
    rawPassword[2] += numList[threadIdx.x];
    rawPassword[3] += numList[threadIdx.y];
    rawPassword[4] += '\0';
    
    char rawEnc[11];
    //Set current ecrypted password
    char *rawEncrypted = cudaEncrypt(rawPassword, rawEnc);

    //Check if input and raw encrypted passwords match and return the output password
    if((comparePasswords(rawEncrypted, inputPassword) == 0)){
        outputPassword[0]= rawPassword[0];
        outputPassword[1]= rawPassword[1];
        outputPassword[2]= rawPassword[2];
        outputPassword[3]= rawPassword[3];
        outputPassword[4]= '\0';
    }
    
}

int main(){

    printf("Encryption Started\n");

    //Size
    int numSize = 10;
    int numSizeMemory = numSize * sizeof(int);

    int letterSize = 26;
    int letterSizeMemory = letterSize * sizeof(int);

    int encPasswordSize = 11;
    int encPasswordSizeMemory = encPasswordSize * sizeof(int);

    int decPasswordSize = 5;
    int decPasswordSizeMemory = decPasswordSize * sizeof(char *);

    //CPU - HOST Variables
    //df77
    char *inputPassword = "fbeice9523";
    char outputPassword[decPasswordSize];

    char numList[numSize];
    char letterList[letterSize];

    int counter = 0;
    
    //creating input values on the CPU
    for(char i = '0'; i <= '9'; i++){
        numList[counter] = i;
        counter++;
    }

    counter = 0;
    for(char i = 'a'; i <= 'z'; i++){
        letterList[counter] = i;
        counter++;
    }
    
    //create GPU variables
    char *deviceLetterList;
    char *deviceNumList;
    char *deviceInputPassword;
    char *deviceOutputPassword;
    
    //allocate memory on the GPU using cudaMalloc
    hipMalloc( (void**) &deviceLetterList, letterSizeMemory);
    hipMalloc( (void**) &deviceNumList, numSizeMemory);
    hipMalloc( (void**) &deviceInputPassword, encPasswordSizeMemory);
    hipMalloc( (void**) &deviceOutputPassword, decPasswordSizeMemory);
    
    //Send to device
    hipMemcpy(deviceLetterList, letterList, letterSizeMemory, hipMemcpyHostToDevice);
    hipMemcpy(deviceNumList, numList, numSizeMemory, hipMemcpyHostToDevice);
    hipMemcpy(deviceInputPassword, inputPassword, encPasswordSizeMemory, hipMemcpyHostToDevice);
    hipMemcpy(deviceOutputPassword, outputPassword, decPasswordSizeMemory, hipMemcpyHostToDevice);
    

    //Blocks and Threads
    dim3 nBlocks = dim3(letterSize, letterSize); 
    dim3 nThreads= dim3(numSize, numSize); 

    cudaDecrypt<<<nBlocks , nThreads>>>(deviceInputPassword, deviceOutputPassword, deviceLetterList, deviceNumList);
    // cudaThreadSynchronize();
    
    hipDeviceSynchronize();
    
    //Get from Device
    hipMemcpy(outputPassword, deviceOutputPassword, decPasswordSizeMemory, hipMemcpyDeviceToHost);

    //Print Output
    printf("Encrypted: %s - > Decrypted: %s\n", inputPassword, outputPassword);

    //Free Cuda Memory
    hipFree(deviceLetterList);
    hipFree(deviceNumList);
    hipFree(deviceInputPassword);
    hipFree(deviceOutputPassword);

    return 0;
}
