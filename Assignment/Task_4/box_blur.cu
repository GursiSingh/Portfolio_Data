#include "hip/hip_runtime.h"
//TASK 4

/*
-   Reading in an image file into a single or 2D array (5 marks)

-   Allocating the correct amount of memory on the GPU based on input data. Memory is freed once used (15 marks)

-   Applying Box filter on image in the kernel function (30 marks)

-   Return blurred image data from the GPU to the CPU (30 marks)

-   Outputting the correct image with Box Blur applied as a file (20 marks)

    Gursimran Singh - 2042387
*/

#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"


//Sum the given center pixel value (R,G,B,A) with all the valid pixels around  
__device__ int sumPixels (int pixel, unsigned char * inputImage, int width , int height){
    int sum = 0;
    int totalPixels =  (width * height * 4) -4;
    //Center

    sum += inputImage[pixel];
    // printf("\n---------------------------\n");
    // printf("CENTER = %d\n", inputImage[pixel]);

    //Top
    if(((pixel - (width*4)) < totalPixels) && ((pixel - (width*4)) >= 0)){
        sum += inputImage[pixel - (width*4)];
        // printf("Pixel = %d Top = %d\n",pixel - (width*4), inputImage[pixel - (width*4)]);
    }

    //Top-Right
    if(((pixel/ 4) +1) < (width*4) && ((((pixel/ 4) +1)% width) != 0)){
        if(((pixel - (width*4) +4) < totalPixels) && ((pixel - (width*4) +4) >= 0) ){
            sum += inputImage[pixel - (width*4) +4];
            
            // printf("Pixel = %d Top-Right = %d\n",(pixel - (width*4) +4), inputImage[pixel - (width*4) +4]);
        }

        //Right
        if(((pixel+4)< totalPixels) && ((pixel+4) > 0)){
            sum += inputImage[pixel+4];
            
            // printf("Pixel = %d Right = %d\n", (pixel+4), inputImage[pixel +4]);
            
        }
    }
    
    //check if is on the left margin
    if(((pixel/4)% width) != 0){
        //Top-Left
        if(((pixel - (width*4) -4) < totalPixels) && ((pixel - (width*4) -4) >= 0) ){
            sum += inputImage[pixel - (width*4) -4];
            
            // printf("Pixel = %d Top-Left = %d\n",(pixel - (width*4) -4), inputImage[pixel - (width*4) -4]);
        }
        //Left
        if(((pixel-4)< totalPixels) && ((pixel-4) >=     0)){
            
            sum += inputImage[pixel-4];
            // printf("Pixel = %d Left = %d\n", (pixel-4), inputImage[pixel -4]);
        }

        //Bottom-Left
        if((((pixel + (width*4)) -4 ) < totalPixels) && (((pixel + (width*4)) -4 ) >= 0) ){
            sum += inputImage[pixel + (width*4) -4];
            // printf("Pixel = %d Bottom-Left = %d\n", ((pixel + (width*4)) -4 ), inputImage[pixel + (width*4) -4]);

        }
        
    }

    //Bottom
    if(((pixel + (width*4))< totalPixels) && ((pixel + (width*4)) >= 0) ){
        sum += inputImage[pixel + (width*4)];
        
        // printf("Pixel = %d Bottom = %d\n", pixel + (width*4), inputImage[pixel + (width*4)]);
    }

    //Bottom-Right
    if(((((pixel/ 4) +1)% width) != 0)){
        if((((pixel + (width*4)) +4 ) < totalPixels) && (((pixel + (width*4)) +4 ) >= 0) ){
            sum += inputImage[pixel + (width*4) +4];
            // printf("Pixel = %d Bottom-Right = %d\n", ((pixel + (width*4)) +4 ), inputImage[pixel + (width*4) +4]);

        }
    }
    



    
    return sum;
}

//This function iterate though all the imagee pixels 4 by 4 if the centeer is foound  it blurs the pixels and return the new image data
__global__ void blurImage(unsigned char * inputImage,unsigned char * outputImage, int centerPixel, int width, int height, int blurredPixels){

    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int pixel = threadID * 4;
    int sumR = 0;
    int sumG = 0;
    int sumB = 0;
    int sumA = 0;
    int totalPixels =  (width * height * 4) -4;

    //center
    if(pixel == centerPixel){
        sumR = sumPixels(centerPixel,inputImage, width, height) / blurredPixels;
        sumG = sumPixels(centerPixel+1,inputImage, width, height) / blurredPixels;
        sumB = sumPixels(centerPixel+2,inputImage, width, height) / blurredPixels;
        sumA = sumPixels(centerPixel+3,inputImage, width, height) / blurredPixels;
        
        //Center
        outputImage[pixel] = sumR;
        outputImage[pixel+1] = sumG;
        outputImage[pixel+2] = sumB;
        outputImage[pixel+3] = sumA;
        
        //Top
        if(((pixel - (width*4)) < totalPixels) && ((pixel - (width*4)) >= 0)){
            
            int top = pixel - (width*4);
            outputImage[top] = sumR;
            outputImage[top+1] = sumG;
            outputImage[top+2] = sumB;
            outputImage[top+3] = sumA;
        }


        //Check if is not right magin
        if(((pixel/ 4) +1) < (width*4) && ((((pixel/ 4) +1)% width) != 0)){

            //Top-Right
            if(((pixel - (width*4) +4) < totalPixels) && ((pixel - (width*4) +4) >= 0) ){
                
                if(((pixel/ 4) +1) < (width*4)){
                    
                    int topRight = pixel - (width*4) +4;
                    outputImage[topRight] = sumR;
                    outputImage[topRight+1] = sumG;
                    outputImage[topRight+2] = sumB;
                    outputImage[topRight+3] = sumA;
                }else{
                    int pix = pixel - (width*4) +4;
                    outputImage[pix] = inputImage[pix];
                    outputImage[pix+1] = inputImage[pix+1];
                    outputImage[pix+2] = inputImage[pix+2];
                    outputImage[pix+3] = inputImage[pix+3];

                }
                
            }

            //Right
            if(((pixel+4)< totalPixels) && ((pixel+4) >= 0)){
                
                int right = pixel+4;
                outputImage[right] = sumR;
                outputImage[right+1] = sumG;
                outputImage[right+2] = sumB;
                outputImage[right+3] = sumA;
            }

            //Bottom-Right
            if((((pixel + (width*4)) +4 ) < totalPixels) && (((pixel + (width*4)) +4 ) >= 0) ){
               
                int bottomRight = pixel + (width*4) +4;
                outputImage[bottomRight] = sumR;
                outputImage[bottomRight+1] = sumG;
                outputImage[bottomRight+2] = sumB;
                outputImage[bottomRight+3] = sumA;
            }
        }else{
            int pixA = pixel - (width*4) +4;
            outputImage[pixA] = inputImage[pixA];
            outputImage[pixA+1] = inputImage[pixA+1];
            outputImage[pixA+2] = inputImage[pixA+2];
            outputImage[pixA+3] = inputImage[pixA+3];

            int pixB =pixel+4;
            outputImage[pixB] = inputImage[pixB];
            outputImage[pixB+1] = inputImage[pixB+1];
            outputImage[pixB+2] = inputImage[pixB+2];
            outputImage[pixB+3] = inputImage[pixB+3];

            int pixC = pixel + (width*4) +4;
            outputImage[pixC] = inputImage[pixC];
            outputImage[pixC+1] = inputImage[pixC+1];
            outputImage[pixC+2] = inputImage[pixC+2];
            outputImage[pixC+3] = inputImage[pixC+3];
        }
        

        //Top-Left and not left margin
        if(((pixel - (width*4) -4) < totalPixels) && ((pixel - (width*4) -4) >= 0) && (((pixel/4)% width) != 0)){
            
            int topLeft = pixel - (width*4) -4;
            outputImage[topLeft] = sumR;
            outputImage[topLeft+1] = sumG;
            outputImage[topLeft+2] = sumB;
            outputImage[topLeft+3] = sumA;
    
        }else{
            int pix = pixel - (width*4) -4;
            outputImage[pix] = inputImage[pix];
            outputImage[pix+1] = inputImage[pix+1];
            outputImage[pix+2] = inputImage[pix+2];
            outputImage[pix+3] = inputImage[pix+3];
        }

        //Bottom
        if(((pixel + (width*4))<= totalPixels) && ((pixel + (width*4)) >= 0) ){
            
            int bottom = pixel + (width*4);
            outputImage[bottom] = sumR;
            outputImage[bottom+1] = sumG;
            outputImage[bottom+2] = sumB;
            outputImage[bottom+3] = sumA;
        }

        //Bottom-Left and not left margin
        if((((pixel + (width*4)) -4 ) < totalPixels) && (((pixel + (width*4)) -4 ) >= 0) && (((pixel/4)% width) != 0)){
            
            int bottomLeft = pixel + (width*4) -4;
            outputImage[bottomLeft] = sumR;
            outputImage[bottomLeft+1] = sumG;
            outputImage[bottomLeft+2] = sumB;
            outputImage[bottomLeft+3] = sumA;
        }else{
            int pix = pixel + (width*4) -4;
            outputImage[pix] = inputImage[pix];
            outputImage[pix+1] = inputImage[pix+1];
            outputImage[pix+2] = inputImage[pix+2];
            outputImage[pix+3] = inputImage[pix+3];
        }

        //Left and not left margin
        if(((pixel-4)< totalPixels) && ((pixel-4) >= 0) && (((pixel/4)% width) != 0)){
            
            int left = pixel-4;
            outputImage[left] = sumR;
            outputImage[left+1] = sumG;
            outputImage[left+2] = sumB;
            outputImage[left+3] = sumA;
            
        }else{
            int pix = pixel -4;
            outputImage[pix] = inputImage[pix];
            outputImage[pix+1] = inputImage[pix+1];
            outputImage[pix+2] = inputImage[pix+2];
            outputImage[pix+3] = inputImage[pix+3];
        }

        
    }

    //Left
    else if(pixel == centerPixel-4){

    }

    //Right
    else if(pixel == centerPixel+4){

    }

    //Top
    else if(pixel == (centerPixel - (width*4))){

    }
    
    //Top-Right
    else if(pixel == (centerPixel - (width*4) +4)){

    }

    //Top-Left
    else if(pixel == (centerPixel - (width*4) -4)){
    }

    //Bottom
    else if(pixel == (centerPixel + (width*4))){

    }


    //Bottom-Right
    else if(pixel == (centerPixel + (width*4) +4)){

    }

    //Bottom-Left
    else if(pixel == (centerPixel + (width*4) -4)){

    }else{
        //If is not the centers
        outputImage[pixel] = inputImage[pixel];
        outputImage[pixel+1] = inputImage[pixel+1];
        outputImage[pixel+2] = inputImage[pixel+2];
        outputImage[pixel+3] = inputImage[pixel+3];
    }
    __syncthreads();

    
}

//Print the image array in the console
void printImage(unsigned char** image2D, unsigned int width, unsigned int height){
    for(int row1=0; row1<height; row1++){
        for(int col1=0; col1<width*4; col1++){
            printf("%d ", image2D[row1][col1]);if((col1+1)%4 == 0);
            printf("| ");
        
        }
        printf("\n");
    }
}

//Return the number of pixels that can be blurred
int getBlurredNum(int width, int height, int row, int col){

    int totalBox = 1;
    
    //Right
    if(row+1 < width){
        totalBox++;
        //Top-Right
        if(col-1 < height && col-1 >= 0){
            totalBox++;
        }
        //Bottom-Right
        if(col+1 < height){
            totalBox++;
        }
    }


    //Bottom
    if(col+1 < height){
        totalBox++;
    }

    //Top
    if(col-1 < height && col-1 >= 0){
        totalBox++;
    }

    //Left
    if(row-1 < width && row-1 >= 0){
        totalBox++;

        //Top-Left
        if(col-1 < height && col-1 >= 0){
            totalBox++;
        }

        //Bottom-Left
        if(col+1 < height){
            totalBox++;
        }
    }
    
    return totalBox;
}

int main(int argc, char *argv[]){

    unsigned int error;
    unsigned char* image;
    unsigned char** image2D;
    unsigned int width;
    unsigned int height;
    const char* filename = argv[1];
    const char* newFilename = "output.png";

    //box filter center
    int centerRow = 3;
    int centerCol = 1;

    //Get image
    error = lodepng_decode32_file(&image, &width, &height, filename);

    if(error){
        printf("error %u: %s\n", error, lodepng_error_text(error));
    }

    //Array Size and memory
    int arraySize = width*height*4;
	int memorySize = arraySize * sizeof(unsigned char);

    unsigned char outputImage[arraySize];

    //Check how many pixeels will be blurred
    int blurredNum = getBlurredNum(width, height, centerRow, centerCol);
    int centerPixel = (width  * (centerCol + 1) - (width - (centerRow))) *4;

    
    //Allocate Memory for rows in the 2D array
    image2D = (unsigned char**)malloc(height * sizeof(unsigned char*));

    //Allocate Memory for cols
    for (int i = 0; i < height; ++i) {
        image2D[i] = (unsigned char*)malloc((width*4) * sizeof(unsigned int));
    }

    //Convert array to 2D in the 2D array
    for(int row=0; row<height; row++){
        for(int col=0; col<width*4; col=col+4){
            image2D[row][col]=image[(row*width*4)+col];
            image2D[row][col+1]=image[(row*width*4)+col+1];
            image2D[row][col+2]=image[(row*width*4)+col+2];
            image2D[row][col+3]=image[(row*width*4)+col+3];
        }
    }
    
    //Cuda Variables
    unsigned char* deviceOutputImage;
    unsigned char* deviceInputImage;

    //Cuda Malloc
	hipMalloc( (void**) &deviceOutputImage, memorySize);
	hipMalloc( (void**) &deviceInputImage, memorySize);

    //Send to device
	hipMemcpy(deviceInputImage, image, memorySize, hipMemcpyHostToDevice);

    //Dimensions
    dim3 nThreads(width, 1, 1);
    dim3 nBlocks(height, 1, 1);

    //Blur Image
    blurImage<<< nBlocks, nThreads >>>(deviceInputImage, deviceOutputImage, centerPixel, width, height, blurredNum);
	hipDeviceSynchronize();

	//Get from Device
	hipMemcpy(outputImage, deviceOutputImage, memorySize, hipMemcpyDeviceToHost);
	
    //Output Image
    //printImage(image2D, width, height);
	lodepng_encode32_file(newFilename, outputImage, width, height);
    printf("\n##### Picture saved as output.png #####\n");   

    //Free memory		
    for (int i = 0; i < height; ++i) {
        free(image2D[i]);
    }
    free(image2D);
    hipFree(deviceOutputImage);
    hipFree(deviceInputImage);

    return 0;
}